#include "hip/hip_runtime.h"
/* For IDE: */
#ifndef __HIPCC__
#define __HIPCC__
#endif

#include "crypto/argon2gpu/cuda/cuda-exception.h"
#include "crypto/argon2gpu/cuda/kernels.h"

#include <stdexcept>
#ifndef NDEBUG
#include <iostream>
#endif

#define ARGON2_D 0
#define ARGON2_I 1
#define ARGON2_ID 2

#define ARGON2_VERSION_10 0x10
#define ARGON2_VERSION_13 0x13

#define ARGON2_BLOCK_SIZE 1024
#define ARGON2_QWORDS_IN_BLOCK (ARGON2_BLOCK_SIZE / 8)
#define ARGON2_SYNC_POINTS 4

#define THREADS_PER_LANE 32
#define QWORDS_PER_THREAD (ARGON2_QWORDS_IN_BLOCK / 32)

namespace argon2gpu
{
namespace cuda
{
using namespace std;

__device__ uint64_t u64_build(uint32_t hi, uint32_t lo)
{
    return ((uint64_t)hi << 32) | (uint64_t)lo;
}

__device__ uint32_t u64_lo(uint64_t x)
{
    return (uint32_t)x;
}

__device__ uint32_t u64_hi(uint64_t x)
{
    return (uint32_t)(x >> 32);
}

__device__ uint64_t u64_shuffle(uint64_t v, uint32_t thread)
{
    uint32_t lo = u64_lo(v);
    uint32_t hi = u64_hi(v);
    lo = __shfl(lo, thread);
    hi = __shfl(hi, thread);
    return u64_build(hi, lo);
}

struct block_g {
    uint64_t data[ARGON2_QWORDS_IN_BLOCK];
};

struct block_th {
    uint64_t a, b, c, d;
};

__device__ uint64_t cmpeq_mask(uint32_t test, uint32_t ref)
{
    uint32_t x = -(uint32_t)(test == ref);
    return u64_build(x, x);
}

__device__ uint64_t block_th_get(const struct block_th* b, uint32_t idx)
{
    uint64_t res = 0;
    res ^= cmpeq_mask(idx, 0) & b->a;
    res ^= cmpeq_mask(idx, 1) & b->b;
    res ^= cmpeq_mask(idx, 2) & b->c;
    res ^= cmpeq_mask(idx, 3) & b->d;
    return res;
}

__device__ void block_th_set(struct block_th* b, uint32_t idx, uint64_t v)
{
    b->a ^= cmpeq_mask(idx, 0) & (v ^ b->a);
    b->b ^= cmpeq_mask(idx, 1) & (v ^ b->b);
    b->c ^= cmpeq_mask(idx, 2) & (v ^ b->c);
    b->d ^= cmpeq_mask(idx, 3) & (v ^ b->d);
}

__device__ void move_block(struct block_th* dst, const struct block_th* src)
{
    *dst = *src;
}

__device__ void xor_block(struct block_th* dst, const struct block_th* src)
{
    dst->a ^= src->a;
    dst->b ^= src->b;
    dst->c ^= src->c;
    dst->d ^= src->d;
}

__device__ void load_block(struct block_th* dst, const struct block_g* src, uint32_t thread)
{
    dst->a = src->data[0 * THREADS_PER_LANE + thread];
    dst->b = src->data[1 * THREADS_PER_LANE + thread];
    dst->c = src->data[2 * THREADS_PER_LANE + thread];
    dst->d = src->data[3 * THREADS_PER_LANE + thread];
}

__device__ void load_block_xor(struct block_th* dst, const struct block_g* src, uint32_t thread)
{
    dst->a ^= src->data[0 * THREADS_PER_LANE + thread];
    dst->b ^= src->data[1 * THREADS_PER_LANE + thread];
    dst->c ^= src->data[2 * THREADS_PER_LANE + thread];
    dst->d ^= src->data[3 * THREADS_PER_LANE + thread];
}

__device__ void store_block(struct block_g* dst, const struct block_th* src, uint32_t thread)
{
    dst->data[0 * THREADS_PER_LANE + thread] = src->a;
    dst->data[1 * THREADS_PER_LANE + thread] = src->b;
    dst->data[2 * THREADS_PER_LANE + thread] = src->c;
    dst->data[3 * THREADS_PER_LANE + thread] = src->d;
}

__device__ uint64_t rotr64(uint64_t x, uint32_t n)
{
    return (x >> n) | (x << (64 - n));
}

__device__ uint64_t f(uint64_t x, uint64_t y)
{
    uint32_t xlo = u64_lo(x);
    uint32_t ylo = u64_lo(y);
    return x + y + 2 * u64_build(__umulhi(xlo, ylo), xlo * ylo);
}

__device__ void g(struct block_th* block)
{
    uint64_t a, b, c, d;
    a = block->a;
    b = block->b;
    c = block->c;
    d = block->d;

    a = f(a, b);
    d = rotr64(d ^ a, 32);
    c = f(c, d);
    b = rotr64(b ^ c, 24);
    a = f(a, b);
    d = rotr64(d ^ a, 16);
    c = f(c, d);
    b = rotr64(b ^ c, 63);

    block->a = a;
    block->b = b;
    block->c = c;
    block->d = d;
}

template <class shuffle>
__device__ void apply_shuffle(struct block_th* block, uint32_t thread)
{
    for (uint32_t i = 0; i < QWORDS_PER_THREAD; i++) {
        uint32_t src_thr = shuffle::apply(thread, i);

        uint64_t v = block_th_get(block, i);
        v = u64_shuffle(v, src_thr);
        block_th_set(block, i, v);
    }
}

__device__ void transpose(struct block_th* block, uint32_t thread)
{
    uint32_t thread_group = (thread & 0x0C) >> 2;
    for (uint32_t i = 1; i < QWORDS_PER_THREAD; i++) {
        uint32_t thr = (i << 2) ^ thread;
        uint32_t idx = thread_group ^ i;

        uint64_t v = block_th_get(block, idx);
        v = u64_shuffle(v, thr);
        block_th_set(block, idx, v);
    }
}

struct identity_shuffle {
    __device__ static uint32_t apply(uint32_t thread, uint32_t idx)
    {
        return thread;
    }
};

struct shift1_shuffle {
    __device__ static uint32_t apply(uint32_t thread, uint32_t idx)
    {
        return (thread & 0x1c) | ((thread + idx) & 0x3);
    }
};

struct unshift1_shuffle {
    __device__ static uint32_t apply(uint32_t thread, uint32_t idx)
    {
        idx = (QWORDS_PER_THREAD - idx) % QWORDS_PER_THREAD;

        return (thread & 0x1c) | ((thread + idx) & 0x3);
    }
};

struct shift2_shuffle {
    __device__ static uint32_t apply(uint32_t thread, uint32_t idx)
    {
        uint32_t lo = (thread & 0x1) | ((thread & 0x10) >> 3);
        lo = (lo + idx) & 0x3;
        return ((lo & 0x2) << 3) | (thread & 0xe) | (lo & 0x1);
    }
};

struct unshift2_shuffle {
    __device__ static uint32_t apply(uint32_t thread, uint32_t idx)
    {
        idx = (QWORDS_PER_THREAD - idx) % QWORDS_PER_THREAD;

        uint32_t lo = (thread & 0x1) | ((thread & 0x10) >> 3);
        lo = (lo + idx) & 0x3;
        return ((lo & 0x2) << 3) | (thread & 0xe) | (lo & 0x1);
    }
};

__device__ void shuffle_block(struct block_th* block, uint32_t thread)
{
    transpose(block, thread);

    g(block);

    apply_shuffle<shift1_shuffle>(block, thread);

    g(block);

    apply_shuffle<unshift1_shuffle>(block, thread);
    transpose(block, thread);

    g(block);

    apply_shuffle<shift2_shuffle>(block, thread);

    g(block);

    apply_shuffle<unshift2_shuffle>(block, thread);
}

__device__ void next_addresses(struct block_th* addr, struct block_th* tmp, uint32_t thread_input, uint32_t thread)
{
    addr->a = u64_build(0, thread_input);
    addr->b = 0;
    addr->c = 0;
    addr->d = 0;

    shuffle_block(addr, thread);

    addr->a ^= u64_build(0, thread_input);
    move_block(tmp, addr);

    shuffle_block(addr, thread);

    xor_block(addr, tmp);
}

__device__ void compute_ref_pos(
    uint32_t lanes,
    uint32_t segment_blocks,
    uint32_t pass,
    uint32_t lane,
    uint32_t slice,
    uint32_t offset,
    uint32_t* ref_lane,
    uint32_t* ref_index)
{
    uint32_t lane_blocks = ARGON2_SYNC_POINTS * segment_blocks;

    *ref_lane = *ref_lane % lanes;

    uint32_t base;
    if (pass != 0) {
        base = lane_blocks - segment_blocks;
    } else {
        if (slice == 0) {
            *ref_lane = lane;
        }
        base = slice * segment_blocks;
    }

    uint32_t ref_area_size = base + offset - 1;
    if (*ref_lane != lane) {
        ref_area_size = min(ref_area_size, base);
    }

    *ref_index = __umulhi(*ref_index, *ref_index);
    *ref_index = ref_area_size - 1 - __umulhi(ref_area_size, *ref_index);

    if (pass != 0 && slice != ARGON2_SYNC_POINTS - 1) {
        *ref_index += (slice + 1) * segment_blocks;
        if (*ref_index >= lane_blocks) {
            *ref_index -= lane_blocks;
        }
    }
}

struct ref {
    uint32_t ref_lane;
    uint32_t ref_index;
};

/*
 * Refs hierarchy:
 * lanes -> passes -> slices -> blocks
 */
template <uint32_t type>
__global__ void argon2_precompute_kernel(
    struct ref* refs,
    uint32_t passes,
    uint32_t lanes,
    uint32_t segment_blocks)
{
    uint32_t block_id = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t thread = threadIdx.x;

    uint32_t segment_addr_blocks = (segment_blocks + ARGON2_QWORDS_IN_BLOCK - 1) / ARGON2_QWORDS_IN_BLOCK;
    uint32_t block = block_id % segment_addr_blocks;
    uint32_t segment = block_id / segment_addr_blocks;

    uint32_t slice, pass, pass_id, lane;
    if (type == ARGON2_ID) {
        slice = segment % (ARGON2_SYNC_POINTS / 2);
        lane = segment / (ARGON2_SYNC_POINTS / 2);
        pass_id = pass = 0;
    } else {
        slice = segment % ARGON2_SYNC_POINTS;
        pass_id = segment / ARGON2_SYNC_POINTS;

        pass = pass_id % passes;
        lane = pass_id / passes;
    }

    struct block_th addr, tmp;

    uint32_t thread_input;
    switch (thread) {
    case 0:
        thread_input = pass;
        break;
    case 1:
        thread_input = lane;
        break;
    case 2:
        thread_input = slice;
        break;
    case 3:
        thread_input = lanes * segment_blocks * ARGON2_SYNC_POINTS;
        break;
    case 4:
        thread_input = passes;
        break;
    case 5:
        thread_input = type;
        break;
    case 6:
        thread_input = block + 1;
        break;
    default:
        thread_input = 0;
        break;
    }

    next_addresses(&addr, &tmp, thread_input, thread);

    refs += segment * segment_blocks;

    for (uint32_t i = 0; i < QWORDS_PER_THREAD; i++) {
        uint32_t pos = i * THREADS_PER_LANE + thread;
        uint32_t offset = block * ARGON2_QWORDS_IN_BLOCK + pos;
        if (offset < segment_blocks) {
            uint64_t v = block_th_get(&addr, i);
            uint32_t ref_index = u64_lo(v);
            uint32_t ref_lane = u64_hi(v);

            compute_ref_pos(lanes, segment_blocks, pass, lane, slice, offset,
                &ref_lane, &ref_index);

            refs[offset].ref_index = ref_index;
            refs[offset].ref_lane = ref_lane;
        }
    }
}

template <uint32_t version>
__device__ void argon2_core(
    struct block_g* memory,
    struct block_g* mem_curr,
    struct block_th* prev,
    struct block_th* tmp,
    uint32_t lanes,
    uint32_t thread,
    uint32_t pass,
    uint32_t ref_index,
    uint32_t ref_lane)
{
    struct block_g* mem_ref = memory + ref_index * lanes + ref_lane;

    if (version != ARGON2_VERSION_10 && pass != 0) {
        load_block(tmp, mem_curr, thread);
        load_block_xor(prev, mem_ref, thread);
        xor_block(tmp, prev);
    } else {
        load_block_xor(prev, mem_ref, thread);
        move_block(tmp, prev);
    }

    shuffle_block(prev, thread);

    xor_block(prev, tmp);

    store_block(mem_curr, prev, thread);
}

template <uint32_t type, uint32_t version>
__device__ void argon2_step_precompute(
    struct block_g* memory,
    struct block_g* mem_curr,
    struct block_th* prev,
    struct block_th* tmp,
    const struct ref** refs,
    uint32_t lanes,
    uint32_t segment_blocks,
    uint32_t thread,
    uint32_t lane,
    uint32_t pass,
    uint32_t slice,
    uint32_t offset)
{
    uint32_t ref_index, ref_lane;
    if (type == ARGON2_I || (type == ARGON2_ID && pass == 0 &&
                                slice < ARGON2_SYNC_POINTS / 2)) {
        ref_index = (*refs)->ref_index;
        ref_lane = (*refs)->ref_lane;
        (*refs)++;
    } else {
        uint64_t v = u64_shuffle(prev->a, 0);
        ref_index = u64_lo(v);
        ref_lane = u64_hi(v);

        compute_ref_pos(lanes, segment_blocks, pass, lane, slice, offset,
            &ref_lane, &ref_index);
    }

    argon2_core<version>(memory, mem_curr, prev, tmp, lanes, thread, pass,
        ref_index, ref_lane);
}

template <uint32_t type, uint32_t version>
__global__ void argon2_kernel_segment_precompute(
    struct block_g* memory,
    const struct ref* refs,
    uint32_t passes,
    uint32_t lanes,
    uint32_t segment_blocks,
    uint32_t pass,
    uint32_t slice)
{
    uint32_t job_id = blockIdx.z * blockDim.z + threadIdx.z;
    uint32_t lane = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t thread = threadIdx.x;

    uint32_t lane_blocks = ARGON2_SYNC_POINTS * segment_blocks;

    /* select job's memory region: */
    memory += (size_t)job_id * lanes * lane_blocks;

    struct block_th prev, tmp;

    struct block_g* mem_segment =
        memory + slice * segment_blocks * lanes + lane;
    struct block_g *mem_prev, *mem_curr;
    uint32_t start_offset = 0;
    if (pass == 0) {
        if (slice == 0) {
            mem_prev = mem_segment + 1 * lanes;
            mem_curr = mem_segment + 2 * lanes;
            start_offset = 2;
        } else {
            mem_prev = mem_segment - lanes;
            mem_curr = mem_segment;
        }
    } else {
        mem_prev = mem_segment + (slice == 0 ? lane_blocks * lanes : 0) - lanes;
        mem_curr = mem_segment;
    }

    load_block(&prev, mem_prev, thread);

    if (type == ARGON2_ID) {
        if (pass == 0 && slice < ARGON2_SYNC_POINTS / 2) {
            refs += lane * (lane_blocks / 2) + slice * segment_blocks;
            refs += start_offset;
        }
    } else {
        refs += (lane * passes + pass) * lane_blocks + slice * segment_blocks;
        refs += start_offset;
    }

    for (uint32_t offset = start_offset; offset < segment_blocks; ++offset) {
        argon2_step_precompute<type, version>(
            memory, mem_curr, &prev, &tmp, &refs, lanes, segment_blocks,
            thread, lane, pass, slice, offset);

        mem_curr += lanes;
    }
}

template <uint32_t type, uint32_t version>
__global__ void argon2_kernel_oneshot_precompute(
    struct block_g* memory,
    const struct ref* refs,
    uint32_t passes,
    uint32_t lanes,
    uint32_t segment_blocks)
{
    uint32_t job_id = blockIdx.z * blockDim.z + threadIdx.z;
    uint32_t lane = threadIdx.y;
    uint32_t thread = threadIdx.x;

    uint32_t lane_blocks = ARGON2_SYNC_POINTS * segment_blocks;

    /* select job's memory region: */
    memory += (size_t)job_id * lanes * lane_blocks;

    struct block_th prev, tmp;

    struct block_g* mem_lane = memory + lane;
    struct block_g* mem_prev = mem_lane + 1 * lanes;
    struct block_g* mem_curr = mem_lane + 2 * lanes;

    load_block(&prev, mem_prev, thread);

    if (type == ARGON2_ID) {
        refs += lane * (lane_blocks / 2) + 2;
    } else {
        refs += lane * passes * lane_blocks + 2;
    }

    uint32_t skip = 2;
    for (uint32_t pass = 0; pass < passes; ++pass) {
        for (uint32_t slice = 0; slice < ARGON2_SYNC_POINTS; ++slice) {
            for (uint32_t offset = 0; offset < segment_blocks; ++offset) {
                if (skip > 0) {
                    --skip;
                    continue;
                }

                argon2_step_precompute<type, version>(
                    memory, mem_curr, &prev, &tmp, &refs, lanes,
                    segment_blocks, thread, lane, pass, slice, offset);

                mem_curr += lanes;
            }

            __syncthreads();
        }

        mem_curr = mem_lane;
    }
}

template <uint32_t type, uint32_t version>
__device__ void argon2_step(
    struct block_g* memory,
    struct block_g* mem_curr,
    struct block_th* prev,
    struct block_th* tmp,
    struct block_th* addr,
    uint32_t lanes,
    uint32_t segment_blocks,
    uint32_t thread,
    uint32_t* thread_input,
    uint32_t lane,
    uint32_t pass,
    uint32_t slice,
    uint32_t offset)
{
    uint32_t ref_index, ref_lane;

    if (type == ARGON2_I || (type == ARGON2_ID && pass == 0 &&
                                slice < ARGON2_SYNC_POINTS / 2)) {
        uint32_t addr_index = offset % ARGON2_QWORDS_IN_BLOCK;
        if (addr_index == 0) {
            if (thread == 6) {
                ++*thread_input;
            }
            next_addresses(addr, tmp, *thread_input, thread);
        }

        uint32_t thr = addr_index % THREADS_PER_LANE;
        uint32_t idx = addr_index / THREADS_PER_LANE;

        uint64_t v = block_th_get(addr, idx);
        v = u64_shuffle(v, thr);
        ref_index = u64_lo(v);
        ref_lane = u64_hi(v);
    } else {
        uint64_t v = u64_shuffle(prev->a, 0);
        ref_index = u64_lo(v);
        ref_lane = u64_hi(v);
    }

    compute_ref_pos(lanes, segment_blocks, pass, lane, slice, offset,
        &ref_lane, &ref_index);

    argon2_core<version>(memory, mem_curr, prev, tmp, lanes, thread, pass,
        ref_index, ref_lane);
}

template <uint32_t type, uint32_t version>
__global__ void argon2_kernel_segment(
    struct block_g* memory,
    uint32_t passes,
    uint32_t lanes,
    uint32_t segment_blocks,
    uint32_t pass,
    uint32_t slice)
{
    uint32_t job_id = blockIdx.z * blockDim.z + threadIdx.z;
    uint32_t lane = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t thread = threadIdx.x;

    uint32_t lane_blocks = ARGON2_SYNC_POINTS * segment_blocks;

    /* select job's memory region: */
    memory += (size_t)job_id * lanes * lane_blocks;

    struct block_th prev, addr, tmp;
    uint32_t thread_input;

    if (type == ARGON2_I || type == ARGON2_ID) {
        switch (thread) {
        case 0:
            thread_input = pass;
            break;
        case 1:
            thread_input = lane;
            break;
        case 2:
            thread_input = slice;
            break;
        case 3:
            thread_input = lanes * lane_blocks;
            break;
        case 4:
            thread_input = passes;
            break;
        case 5:
            thread_input = type;
            break;
        default:
            thread_input = 0;
            break;
        }

        if (pass == 0 && slice == 0 && segment_blocks > 2) {
            if (thread == 6) {
                ++thread_input;
            }
            next_addresses(&addr, &tmp, thread_input, thread);
        }
    }

    struct block_g* mem_segment =
        memory + slice * segment_blocks * lanes + lane;
    struct block_g *mem_prev, *mem_curr;
    uint32_t start_offset = 0;
    if (pass == 0) {
        if (slice == 0) {
            mem_prev = mem_segment + 1 * lanes;
            mem_curr = mem_segment + 2 * lanes;
            start_offset = 2;
        } else {
            mem_prev = mem_segment - lanes;
            mem_curr = mem_segment;
        }
    } else {
        mem_prev = mem_segment + (slice == 0 ? lane_blocks * lanes : 0) - lanes;
        mem_curr = mem_segment;
    }

    load_block(&prev, mem_prev, thread);

    for (uint32_t offset = start_offset; offset < segment_blocks; ++offset) {
        argon2_step<type, version>(
            memory, mem_curr, &prev, &tmp, &addr, lanes, segment_blocks,
            thread, &thread_input, lane, pass, slice, offset);

        mem_curr += lanes;
    }
}

template <uint32_t type, uint32_t version>
__global__ void argon2_kernel_oneshot(
    struct block_g* memory,
    uint32_t passes,
    uint32_t lanes,
    uint32_t segment_blocks)
{
    uint32_t job_id = blockIdx.z * blockDim.z + threadIdx.z;
    uint32_t lane = threadIdx.y;
    uint32_t thread = threadIdx.x;

    uint32_t lane_blocks = ARGON2_SYNC_POINTS * segment_blocks;

    /* select job's memory region: */
    memory += (size_t)job_id * lanes * lane_blocks;

    struct block_th prev, addr, tmp;
    uint32_t thread_input;

    if (type == ARGON2_I || type == ARGON2_ID) {
        switch (thread) {
        case 1:
            thread_input = lane;
            break;
        case 3:
            thread_input = lanes * lane_blocks;
            break;
        case 4:
            thread_input = passes;
            break;
        case 5:
            thread_input = type;
            break;
        default:
            thread_input = 0;
            break;
        }

        if (segment_blocks > 2) {
            if (thread == 6) {
                ++thread_input;
            }
            next_addresses(&addr, &tmp, thread_input, thread);
        }
    }

    struct block_g* mem_lane = memory + lane;
    struct block_g* mem_prev = mem_lane + 1 * lanes;
    struct block_g* mem_curr = mem_lane + 2 * lanes;

    load_block(&prev, mem_prev, thread);

    uint32_t skip = 2;
    for (uint32_t pass = 0; pass < passes; ++pass) {
        for (uint32_t slice = 0; slice < ARGON2_SYNC_POINTS; ++slice) {
            for (uint32_t offset = 0; offset < segment_blocks; ++offset) {
                if (skip > 0) {
                    --skip;
                    continue;
                }

                argon2_step<type, version>(
                    memory, mem_curr, &prev, &tmp, &addr, lanes,
                    segment_blocks, thread, &thread_input, lane, pass,
                    slice, offset);

                mem_curr += lanes;
            }

            __syncthreads();

            if (type == ARGON2_I || type == ARGON2_ID) {
                if (thread == 2) {
                    ++thread_input;
                }
                if (thread == 6) {
                    thread_input = 0;
                }
            }
        }
        if (type == ARGON2_I) {
            if (thread == 0) {
                ++thread_input;
            }
            if (thread == 2) {
                thread_input = 0;
            }
        }
        mem_curr = mem_lane;
    }
}


static void setCudaDevice(int deviceIndex)
{
    int currentIndex = -1;
    CudaException::check(hipGetDevice(&currentIndex));
    if (currentIndex != deviceIndex) {
        CudaException::check(hipSetDevice(deviceIndex));
    }
}

KernelRunner::KernelRunner(uint32_t type, uint32_t version, uint32_t passes, uint32_t lanes, uint32_t segmentBlocks, uint32_t batchSize, bool bySegment, bool precompute, int deviceIndex)
    : type(type), version(version), passes(passes), lanes(lanes),
      segmentBlocks(segmentBlocks), batchSize(batchSize), bySegment(bySegment), deviceIndex(deviceIndex),
      precompute(precompute), stream(nullptr), memory(nullptr),
      refs(nullptr), start(nullptr), end(nullptr)
{
    setCudaDevice(deviceIndex);

    // FIXME: check overflow:
    size_t memorySize = static_cast<size_t>(lanes) * segmentBlocks * ARGON2_SYNC_POINTS * ARGON2_BLOCK_SIZE * batchSize;

#ifndef NDEBUG
    std::cerr << "[INFO] Allocating " << memorySize << " bytes for memory..."
              << std::endl;
#endif

    CudaException::check(hipMalloc(&memory, memorySize));

    CudaException::check(hipEventCreate(&start));
    CudaException::check(hipEventCreate(&end));

    CudaException::check(hipStreamCreate(&stream));

    if ((type == ARGON2_I || type == ARGON2_ID) && precompute) {
        uint32_t segments =
            type == ARGON2_ID ? lanes * (ARGON2_SYNC_POINTS / 2) : passes * lanes * ARGON2_SYNC_POINTS;

        size_t refsSize = segments * segmentBlocks * sizeof(struct ref);

#ifndef NDEBUG
        std::cerr << "[INFO] Allocating " << refsSize << " bytes for refs..."
                  << std::endl;
#endif

        CudaException::check(hipMalloc(&refs, refsSize));

        precomputeRefs();
        CudaException::check(hipStreamSynchronize(stream));
    }
}

void KernelRunner::precomputeRefs()
{
    struct ref* refs = (struct ref*)this->refs;

    uint32_t segmentAddrBlocks = (segmentBlocks + ARGON2_QWORDS_IN_BLOCK - 1) / ARGON2_QWORDS_IN_BLOCK;
    uint32_t segments =
        type == ARGON2_ID ? lanes * (ARGON2_SYNC_POINTS / 2) : passes * lanes * ARGON2_SYNC_POINTS;

    dim3 blocks = dim3(1, segments * segmentAddrBlocks);
    dim3 threads = dim3(THREADS_PER_LANE);

    if (type == ARGON2_I) {
        argon2_precompute_kernel<ARGON2_I>
            <<<blocks, threads, 0, stream>>>(
                refs, passes, lanes, segmentBlocks);
    } else {
        argon2_precompute_kernel<ARGON2_ID>
            <<<blocks, threads, 0, stream>>>(
                refs, passes, lanes, segmentBlocks);
    }
}

KernelRunner::~KernelRunner()
{
    if (start != nullptr) {
        hipEventDestroy(start);
    }
    if (end != nullptr) {
        hipEventDestroy(end);
    }
    if (stream != nullptr) {
        hipStreamDestroy(stream);
    }
    if (memory != nullptr) {
        hipFree(memory);
    }
    if (refs != nullptr) {
        hipFree(refs);
    }
}

void KernelRunner::writeInputMemory(uint32_t jobId, const void* buffer)
{
    std::size_t memorySize = static_cast<size_t>(lanes) * segmentBlocks * ARGON2_SYNC_POINTS * ARGON2_BLOCK_SIZE;
    std::size_t size = static_cast<size_t>(lanes) * 2 * ARGON2_BLOCK_SIZE;
    std::size_t offset = memorySize * jobId;
    auto mem = static_cast<uint8_t*>(memory) + offset;
    CudaException::check(hipMemcpyAsync(mem, buffer, size,
        hipMemcpyHostToDevice, stream));
    CudaException::check(hipStreamSynchronize(stream));
}

void KernelRunner::readOutputMemory(uint32_t jobId, void* buffer)
{
    std::size_t memorySize = static_cast<size_t>(lanes) * segmentBlocks * ARGON2_SYNC_POINTS * ARGON2_BLOCK_SIZE;
    std::size_t size = static_cast<size_t>(lanes) * ARGON2_BLOCK_SIZE;
    std::size_t offset = memorySize * (jobId + 1) - size;
    auto mem = static_cast<uint8_t*>(memory) + offset;
    CudaException::check(hipMemcpyAsync(buffer, mem, size,
        hipMemcpyDeviceToHost, stream));
    CudaException::check(hipStreamSynchronize(stream));
}

void KernelRunner::runKernelSegment(uint32_t lanesPerBlock,
    uint32_t jobsPerBlock,
    uint32_t pass,
    uint32_t slice)
{
    if (lanesPerBlock > lanes || lanes % lanesPerBlock != 0) {
        throw std::logic_error("Invalid lanesPerBlock!");
    }

    if (jobsPerBlock > batchSize || batchSize % jobsPerBlock != 0) {
        throw std::logic_error("Invalid jobsPerBlock!");
    }

    struct block_g* memory_blocks = (struct block_g*)memory;
    dim3 blocks = dim3(1, lanes / lanesPerBlock, batchSize / jobsPerBlock);
    dim3 threads = dim3(THREADS_PER_LANE, lanesPerBlock, jobsPerBlock);
    if (type == ARGON2_I) {
        if (precompute) {
            struct ref* refs = (struct ref*)this->refs;
            if (version == ARGON2_VERSION_10) {
                argon2_kernel_segment_precompute<ARGON2_I, ARGON2_VERSION_10>
                    <<<blocks, threads, 0, stream>>>(
                        memory_blocks, refs, passes, lanes, segmentBlocks,
                        pass, slice);
            } else {
                argon2_kernel_segment_precompute<ARGON2_I, ARGON2_VERSION_13>
                    <<<blocks, threads, 0, stream>>>(
                        memory_blocks, refs, passes, lanes, segmentBlocks,
                        pass, slice);
            }
        } else {
            if (version == ARGON2_VERSION_10) {
                argon2_kernel_segment<ARGON2_I, ARGON2_VERSION_10>
                    <<<blocks, threads, 0, stream>>>(
                        memory_blocks, passes, lanes, segmentBlocks,
                        pass, slice);
            } else {
                argon2_kernel_segment<ARGON2_I, ARGON2_VERSION_13>
                    <<<blocks, threads, 0, stream>>>(
                        memory_blocks, passes, lanes, segmentBlocks,
                        pass, slice);
            }
        }
    } else if (type == ARGON2_ID) {
        if (precompute) {
            struct ref* refs = (struct ref*)this->refs;
            if (version == ARGON2_VERSION_10) {
                argon2_kernel_segment_precompute<ARGON2_ID, ARGON2_VERSION_10>
                    <<<blocks, threads, 0, stream>>>(
                        memory_blocks, refs, passes, lanes, segmentBlocks,
                        pass, slice);
            } else {
                argon2_kernel_segment_precompute<ARGON2_ID, ARGON2_VERSION_13>
                    <<<blocks, threads, 0, stream>>>(
                        memory_blocks, refs, passes, lanes, segmentBlocks,
                        pass, slice);
            }
        } else {
            if (version == ARGON2_VERSION_10) {
                argon2_kernel_segment<ARGON2_ID, ARGON2_VERSION_10>
                    <<<blocks, threads, 0, stream>>>(
                        memory_blocks, passes, lanes, segmentBlocks,
                        pass, slice);
            } else {
                argon2_kernel_segment<ARGON2_ID, ARGON2_VERSION_13>
                    <<<blocks, threads, 0, stream>>>(
                        memory_blocks, passes, lanes, segmentBlocks,
                        pass, slice);
            }
        }
    } else {
        if (version == ARGON2_VERSION_10) {
            argon2_kernel_segment<ARGON2_D, ARGON2_VERSION_10>
                <<<blocks, threads, 0, stream>>>(
                    memory_blocks, passes, lanes, segmentBlocks,
                    pass, slice);
        } else {
            argon2_kernel_segment<ARGON2_D, ARGON2_VERSION_13>
                <<<blocks, threads, 0, stream>>>(
                    memory_blocks, passes, lanes, segmentBlocks,
                    pass, slice);
        }
    }
}

void KernelRunner::runKernelOneshot(uint32_t lanesPerBlock,
    uint32_t jobsPerBlock)
{
    if (lanesPerBlock != lanes) {
        throw std::logic_error("Invalid lanesPerBlock!");
    }

    if (jobsPerBlock > batchSize || batchSize % jobsPerBlock != 0) {
        throw std::logic_error("Invalid jobsPerBlock!");
    }

    struct block_g* memory_blocks = (struct block_g*)memory;
    dim3 blocks = dim3(1, 1, batchSize / jobsPerBlock);
    dim3 threads = dim3(THREADS_PER_LANE, lanes, jobsPerBlock);
    if (type == ARGON2_I) {
        if (precompute) {
            struct ref* refs = (struct ref*)this->refs;
            if (version == ARGON2_VERSION_10) {
                argon2_kernel_oneshot_precompute<ARGON2_I, ARGON2_VERSION_10>
                    <<<blocks, threads, 0, stream>>>(
                        memory_blocks, refs, passes, lanes, segmentBlocks);
            } else {
                argon2_kernel_oneshot_precompute<ARGON2_I, ARGON2_VERSION_13>
                    <<<blocks, threads, 0, stream>>>(
                        memory_blocks, refs, passes, lanes, segmentBlocks);
            }
        } else {
            if (version == ARGON2_VERSION_10) {
                argon2_kernel_oneshot<ARGON2_I, ARGON2_VERSION_10>
                    <<<blocks, threads, 0, stream>>>(
                        memory_blocks, passes, lanes, segmentBlocks);
            } else {
                argon2_kernel_oneshot<ARGON2_I, ARGON2_VERSION_13>
                    <<<blocks, threads, 0, stream>>>(
                        memory_blocks, passes, lanes, segmentBlocks);
            }
        }
    } else if (type == ARGON2_ID) {
        if (precompute) {
            struct ref* refs = (struct ref*)this->refs;
            if (version == ARGON2_VERSION_10) {
                argon2_kernel_oneshot_precompute<ARGON2_ID, ARGON2_VERSION_10>
                    <<<blocks, threads, 0, stream>>>(
                        memory_blocks, refs, passes, lanes, segmentBlocks);
            } else {
                argon2_kernel_oneshot_precompute<ARGON2_ID, ARGON2_VERSION_13>
                    <<<blocks, threads, 0, stream>>>(
                        memory_blocks, refs, passes, lanes, segmentBlocks);
            }
        } else {
            if (version == ARGON2_VERSION_10) {
                argon2_kernel_oneshot<ARGON2_ID, ARGON2_VERSION_10>
                    <<<blocks, threads, 0, stream>>>(
                        memory_blocks, passes, lanes, segmentBlocks);
            } else {
                argon2_kernel_oneshot<ARGON2_ID, ARGON2_VERSION_13>
                    <<<blocks, threads, 0, stream>>>(
                        memory_blocks, passes, lanes, segmentBlocks);
            }
        }
    } else {
        if (version == ARGON2_VERSION_10) {
            argon2_kernel_oneshot<ARGON2_D, ARGON2_VERSION_10>
                <<<blocks, threads, 0, stream>>>(
                    memory_blocks, passes, lanes, segmentBlocks);
        } else {
            argon2_kernel_oneshot<ARGON2_D, ARGON2_VERSION_13>
                <<<blocks, threads, 0, stream>>>(
                    memory_blocks, passes, lanes, segmentBlocks);
        }
    }
}

void KernelRunner::run(uint32_t lanesPerBlock, uint32_t jobsPerBlock)
{
    setCudaDevice(deviceIndex);
    CudaException::check(hipEventRecord(start, stream));

    if (bySegment) {
        for (uint32_t pass = 0; pass < passes; pass++) {
            for (uint32_t slice = 0; slice < ARGON2_SYNC_POINTS; slice++) {
                runKernelSegment(lanesPerBlock, jobsPerBlock, pass, slice);
            }
        }
    } else {
        runKernelOneshot(lanesPerBlock, jobsPerBlock);
    }

    CudaException::check(hipGetLastError());

    CudaException::check(hipEventRecord(end, stream));
}

float KernelRunner::finish()
{
    CudaException::check(hipStreamSynchronize(stream));

    float time = 0.0;
    CudaException::check(hipEventElapsedTime(&time, start, end));
    return time;
}

} // namespace cuda
} // namespace argon2gpu
